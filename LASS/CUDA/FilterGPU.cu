#include "hip/hip_runtime.h"

#include "../../LASS/src/LASS.h"
#include "FilterGPU.h"

#define CUDA_CHECK(call)                                                     \
do {                                                                         \
    hipError_t err = call;                                                  \
    if (err != hipSuccess) {                                                \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,     \
                hipGetErrorString(err));                                    \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
} while (0)

#define CEIL_MULT(x, y)  ( (( (x) + (y) - 1 ) / (y) ) * (y) )

__global__ void LPCombFilterGPU(float *inputSample, float *outputSample, float inputGain, long inputDelay, float inputLpf_gain, float *delaybuf0, float *delaybuf1, long sampleSize){
    float gain=inputGain, lpf_gain=inputLpf_gain;
    double gaine;
    int tx = threadIdx.x, idx;
    long delay = inputDelay, ps=(double)(sampleSize + delay - 1)/delay, pb=(double)(delay + blockDim.x - 1)/blockDim.x;
    //__shared__ float Z0[4096], Z1[4096];
    float *Zsrc = delaybuf0, *Zdest = delaybuf1, *Ztemp;

    for (int i = 0; i < pb; i++){
        idx = i * blockDim.x + tx;
        if (idx < delay)
            outputSample[idx] = 0;
    }

    for (int i = 0; i < pb; i++){
        idx = i * blockDim.x + tx;
        if (idx < delay){
            Zsrc[idx] = inputSample[idx];
            outputSample[idx + delay] = Zsrc[idx];
        }
    }

    for(int j = 2; j < ps; ++j){
        gaine = lpf_gain;
        for (int off = 1; off < pb * blockDim.x; off *= 2) {
            __syncthreads();
            for (int i = 0; i < pb; i++) {
                idx = tx + i * blockDim.x;
                if (idx >= off) {
                    Zdest[idx] = Zsrc[idx] + gaine * Zsrc[idx - off];
                }
                else 
                    Zdest[idx] = Zsrc[idx];
            }
            gaine *= gaine;
            Ztemp = Zsrc;
            Zsrc = Zdest;
            Zdest = Ztemp;
        }

        __syncthreads();
        
        for (int i = 0; i < pb; ++i){
            idx = i * blockDim.x + tx;
            if (idx < delay){
                Zsrc[idx] = gain * Zsrc[idx] + inputSample[(j-1) * delay + idx];
                outputSample[j * delay + idx] = Zsrc[idx];
            }
        }
        if (tx == 0)
            Zsrc[0] += outputSample[j * delay - 1];
    }
}

__global__ void HexAllPassFilterGPU(float *inputSample, float *inputSample0, float *inputSample1, float *inputSample2, float *inputSample3, float *inputSample4, float *inputSample5, float* outputSample, float* envData, float inputGain, long inputDelay, float *delaybuf0, float *delaybuf1, long sampleSize){
    float gain=inputGain, gsqrd=gain*gain, x;
    int tx = blockIdx.x*blockDim.x+threadIdx.x, idx;
    long delay = inputDelay, ps=(double)(sampleSize+delay-1)/delay, pb=(double)(delay+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
    int stridesz=(delay+blockDim.x-1)/blockDim.x*blockDim.x;
    //__shared__ float X[4096], Y[4096];

    for (int i = 0; i < pb; ++i){
        idx = i * gridDim.x * blockDim.x + tx;
        if (idx < delay){
            delaybuf0[stridesz*blockIdx.x+blockDim.x*i+threadIdx.x] = (inputSample0[idx]+inputSample1[idx]+inputSample2[idx]+inputSample3[idx]+inputSample4[idx]+inputSample5[idx])/6;
            delaybuf1[stridesz*blockIdx.x+blockDim.x*i+threadIdx.x] = -gain*delaybuf0[stridesz*blockIdx.x+blockDim.x*i+threadIdx.x];
            outputSample[idx] = delaybuf1[stridesz*blockIdx.x+blockDim.x*i+threadIdx.x]*envData[idx] + (1-envData[idx])*inputSample[idx];
        }
    }

    for(int i=1; i<ps; ++i){
        for (int j = 0; j < pb; ++j){
            idx = j* gridDim.x * blockDim.x + tx;
            if (idx < delay&& i*delay+idx < sampleSize){
                x=delaybuf0[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x];
                delaybuf0[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x] = (inputSample0[i*delay+idx]+inputSample1[i*delay+idx]+inputSample2[i*delay+idx]+inputSample3[i*delay+idx]+inputSample4[i*delay+idx]+inputSample5[i*delay+idx])/6;
                delaybuf1[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x] = -gain*delaybuf0[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x]+(1-gsqrd)*(gain*delaybuf1[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x]+x);
                outputSample[i*delay+idx] = delaybuf1[stridesz*blockIdx.x+blockDim.x*j+threadIdx.x]*envData[i*delay+idx]+(1-envData[i*delay+idx])*inputSample[i*delay+idx];
            }
        }
    }
}

__global__ void getEnvData(float *xyPoints, int *segmentTypes, float *envData, int segmentSize, long sampleSize){
    int tx = blockDim.x*blockIdx.x+threadIdx.x, samples, idx, start, i, j;
    float x0, y0, x1, y1, m0=0, m1=0, t, a, b;
    for(i=0; i<segmentSize; ++i){
        x0=xyPoints[2*i];
        y0=xyPoints[2*i+1];
        x1=xyPoints[2*i+2];
        y1=xyPoints[2*i+3];
        samples=(x1-x0)*sampleSize;
        start=sampleSize * x0;
        if(segmentTypes[i]==2){

            m0=(y1-y0)/(x1-x0)/sampleSize;
            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x)+ tx;
                envData[start + idx] = y0 + idx*m0;
            }

            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start+ idx<sampleSize)
                envData[start + idx] = y0 + idx*m0;

        }
        else if(segmentTypes[i]==1){
            //Cubic Hermite spline
            if(i!=0){
                if(segmentTypes[i-1]==0){
                    a=y0>y1?3:-3;
                    b=pow(2.71828, (double)a);
                    m0=a*(y0-xyPoints[2*i-1])*b/(x0-xyPoints[2*i-2])/(b-1);
                }
                else if(segmentTypes[i-1]==1)
                    m0=(y1-xyPoints[2*i-1])/(x1-xyPoints[2*i-2]);
                else 
                    m0=(y0-xyPoints[2*i-1])/(x0-xyPoints[2*i-2]);
            }
            else
                m0=0;

            if(i+1<segmentSize){
                if(segmentTypes[i+1]==0){
                    a=xyPoints[2*i+5]>y1?3:-3;
                    m1=a*(xyPoints[2*i+5]-y1)/(xyPoints[2*i+4]-x1)/(pow(2.71828, (double)a)-1);
                }
                else if(segmentTypes[i+1]==1)
                    m1=(xyPoints[2*i+5]-y0)/(xyPoints[2*i+4]-x0);
                else 
                    m1=(xyPoints[2*i+5]-y1)/(xyPoints[2*i+4]-x1);
            }
            else
                m1=0;

            a= -3*y0+3*y1-2*m0-m1;
            b= 2*y0-2*y1+m0+m1;

            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x) + tx;
                t=(float)idx/samples;
                envData[start + idx] = y0 + m0*t + a*t*t + b*t*t*t;
            }
            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start + idx<sampleSize){
                t=(float)idx/samples;
                envData[start + idx] = y0 + m0*t + a*t*t + b*t*t*t;
            }
        }
        else{
            a=y1>y0?3:-3;
            b=pow(2.71828, (double)a);

            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x) + tx;
                envData[start + idx] = y0+(y1-y0)*(1-pow(2.71828, (double)a*idx/samples))/(1-b);
            }

            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start+ idx<sampleSize)
                envData[start + idx] = y0+(y1-y0)*(1-pow(2.71828, (double)a*idx/samples))/(1-b);
        }
        __syncthreads();
    }
}

void plotWithGnuplot(const std::vector<float>& data) {
    FILE *gnuplotPipe = popen("gnuplot -persistent", "w");
    if (gnuplotPipe) {
        // Set up the plot
        fprintf(gnuplotPipe, "set title 'Plot of Floats from 0 to 2'\n");
        fprintf(gnuplotPipe, "set xlabel 'Index'\n");
        fprintf(gnuplotPipe, "set ylabel 'Value'\n");
        fprintf(gnuplotPipe, "plot '-' with lines\n");
        
        // Send data to GNUplot
        for(size_t i = 0; i < data.size(); ++i){
            fprintf(gnuplotPipe, "%zu %f\n", i, data[i]);
        }
        fprintf(gnuplotPipe, "e\n");
        pclose(gnuplotPipe);
    } else {
        std::cerr << "Could not open pipe to GNUplot.\n";
    }
}

SoundSample* do_reverb_SoundSample_GPU(SoundSample *inWave, Envelope *percentReverbinput, LPCombFilter **lpCombFilter, AllPassFilter *allPassFilter){ 
    SoundSample *outWave=new SoundSample(inWave->getSampleCount(),inWave->getSamplingRate());
    float *inWaveData=inWave->getData(), *outWaveDataD0, *outWaveDataD1, *outWaveDataD2, *outWaveDataD3, *outWaveDataD4, *outWaveDataD5, *outWaveDataD, *inWaveDataD, *outWaveData=new float[inWave->getSampleCount()];
    float *delay0bufD0, *delay0bufD1, *delay0bufD2, *delay0bufD3, *delay0bufD4, *delay0bufD5;
    float *delay1bufD0, *delay1bufD1, *delay1bufD2, *delay1bufD3, *delay1bufD4, *delay1bufD5;
    float *delay0bufAllP, *delay1bufAllP;
    long sampleSize=inWave->getSampleCount();
    float durationofEnv=percentReverbinput->getDuration();
    float *envData=new float[sampleSize], *envDataD, *envXY, *envXYD;
    int *envSegType, *envSegTypeD, segSize;

    Collection<envelope_segment> *segs=percentReverbinput->getSegments();
    envelope_segment seg;
    segSize=segs->size();

    envXY=new float[segSize*2];
    envSegType=new int[segSize-1];

    for (int i = 1; i < segSize; ++i){
        seg=segs->get(i);
        envXY[i*2]=seg.x;
        envXY[i*2+1]=seg.y;
        envSegType[i-1]=seg.interType;
    }
    
    seg=segs->get(0);
    envXY[0]=seg.x;
    envXY[1]=seg.y;

    hipMalloc(&inWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD0, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD1, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD2, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD3, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD4, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD5, sampleSize*sizeof(float));
    hipMalloc(&delay0bufD0, CEIL_MULT(lpCombFilter[0]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay0bufD1, CEIL_MULT(lpCombFilter[1]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay0bufD2, CEIL_MULT(lpCombFilter[2]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay0bufD3, CEIL_MULT(lpCombFilter[3]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay0bufD4, CEIL_MULT(lpCombFilter[4]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay0bufD5, CEIL_MULT(lpCombFilter[5]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD0, CEIL_MULT(lpCombFilter[0]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD1, CEIL_MULT(lpCombFilter[1]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD2, CEIL_MULT(lpCombFilter[2]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD3, CEIL_MULT(lpCombFilter[3]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD4, CEIL_MULT(lpCombFilter[4]->get_D(), 256)*sizeof(float));
    hipMalloc(&delay1bufD5, CEIL_MULT(lpCombFilter[5]->get_D(), 256)*sizeof(float));
    hipMalloc(&outWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&envDataD, sampleSize*sizeof(float));
    hipMemcpy(inWaveDataD, inWaveData, sampleSize*sizeof(float), hipMemcpyHostToDevice);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD0, lpCombFilter[0]->get_g(), lpCombFilter[0]->get_D(), lpCombFilter[0]->get_lpf_g(), delay0bufD0, delay1bufD0, sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD1, lpCombFilter[1]->get_g(), lpCombFilter[1]->get_D(), lpCombFilter[1]->get_lpf_g(), delay0bufD1, delay1bufD1, sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD2, lpCombFilter[2]->get_g(), lpCombFilter[2]->get_D(), lpCombFilter[2]->get_lpf_g(), delay0bufD2, delay1bufD2, sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD3, lpCombFilter[3]->get_g(), lpCombFilter[3]->get_D(), lpCombFilter[3]->get_lpf_g(), delay0bufD3, delay1bufD3, sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD4, lpCombFilter[4]->get_g(), lpCombFilter[4]->get_D(), lpCombFilter[4]->get_lpf_g(), delay0bufD4, delay1bufD4, sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD5, lpCombFilter[5]->get_g(), lpCombFilter[5]->get_D(), lpCombFilter[5]->get_lpf_g(), delay0bufD5, delay1bufD5, sampleSize);
    hipDeviceSynchronize();

    hipMalloc(&envXYD, segSize*2*sizeof(float));
    hipMalloc(&envSegTypeD, (segSize-1)*sizeof(int));
    hipMalloc(&delay0bufAllP, CEIL_MULT(allPassFilter->get_D(), 256)*6*sizeof(float));
    hipMalloc(&delay1bufAllP, CEIL_MULT(allPassFilter->get_D(), 256)*6*sizeof(float));

    hipMemcpy(envXYD, envXY, segSize*2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(envSegTypeD, envSegType, (segSize-1)*sizeof(int), hipMemcpyHostToDevice);

    getEnvData<<<6, 256>>>(envXYD, envSegTypeD, envDataD, segSize-1, sampleSize);
    hipDeviceSynchronize();

    //cout<<"envXY "<<envXY[0]<<" "<<envXY[1]<<" "<<envXY[2]<<" "<<envXY[3]<<" "<<envXY[4]<<" "<<envXY[5]<<endl;
    //cout<<"envSegType "<<envSegType[0]<<" "<<envSegType[1]<<endl;
    //cout<<"segSize "<<segSize<<endl;
    
    //hipMemcpy(envData, envDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    //std::vector<float> plot;
    //
    //for (int i = 0; i < sampleSize; i+=1000) {
    //    plot.push_back(envData[i]);
    //}
    //plotWithGnuplot(plot);

    HexAllPassFilterGPU<<<6, 256>>>(inWaveDataD, outWaveDataD0, outWaveDataD1, outWaveDataD2, outWaveDataD3, outWaveDataD4, outWaveDataD5, outWaveDataD, envDataD, allPassFilter->get_g(), allPassFilter->get_D(), delay0bufAllP, delay1bufAllP, sampleSize);

    hipDeviceSynchronize();


    hipMemcpy(outWave->getData(), outWaveDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<"outwave 0 "<<(*outWave)[0]<<endl;
    // cout<<"outwave 1000 "<<(*outWave)[1000]<<endl;
    // cout<<"outwave 10000 "<<(*outWave)[10000]<<endl;
    // cout<<"outwave 100000 "<<(*outWave)[100000]<<endl;

    hipFree(inWaveDataD);
    hipFree(outWaveDataD0);
    hipFree(outWaveDataD1);
    hipFree(outWaveDataD2);
    hipFree(outWaveDataD3);
    hipFree(outWaveDataD4);
    hipFree(outWaveDataD5);
    hipFree(delay0bufD0);
    hipFree(delay0bufD1);
    hipFree(delay0bufD2);
    hipFree(delay0bufD3);
    hipFree(delay0bufD4);
    hipFree(delay0bufD5);
    hipFree(delay1bufD0);
    hipFree(delay1bufD1);
    hipFree(delay1bufD2);
    hipFree(delay1bufD3);
    hipFree(delay1bufD4);
    hipFree(delay1bufD5);
    hipFree(delay0bufAllP);
    hipFree(delay1bufAllP);
    hipFree(outWaveDataD);
    hipFree(envDataD);
    hipFree(envXYD);
    hipFree(envSegTypeD);
    delete[] envData;
    delete[] outWaveData;
    delete[] envXY;
    delete[] envSegType;

    return outWave;
}